/**
 * SPDX-FileCopyrightText: Copyright (c) 2021-2023, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "morpheus/utilities/type_util.hpp"

#include <stdexcept>
#include <string>

namespace morpheus {

DType::DType(const DataType& dtype) : DataType(dtype.type_id()) {}
DType::DType(TypeId tid) : DataType(tid) {}

// Cudf representation
cudf::type_id DType::cudf_type_id() const
{
    switch (m_type_id)
    {
    case TypeId::INT8:
        return cudf::type_id::INT8;
    case TypeId::INT16:
        return cudf::type_id::INT16;
    case TypeId::INT32:
        return cudf::type_id::INT32;
    case TypeId::INT64:
        return cudf::type_id::INT64;
    case TypeId::UINT8:
        return cudf::type_id::UINT8;
    case TypeId::UINT16:
        return cudf::type_id::UINT16;
    case TypeId::UINT32:
        return cudf::type_id::UINT32;
    case TypeId::UINT64:
        return cudf::type_id::UINT64;
    case TypeId::FLOAT32:
        return cudf::type_id::FLOAT32;
    case TypeId::FLOAT64:
        return cudf::type_id::FLOAT64;
    case TypeId::BOOL8:
        return cudf::type_id::BOOL8;
    case TypeId::EMPTY:
    case TypeId::NUM_TYPE_IDS:
    default:
        throw std::runtime_error("Not supported");
    }
}

// Returns the triton string representation
std::string DType::triton_str() const
{
    // Triton doesn't have any definitions or enums. Wow
    switch (m_type_id)
    {
    case TypeId::INT8:
        return "INT8";
    case TypeId::INT16:
        return "INT16";
    case TypeId::INT32:
        return "INT32";
    case TypeId::INT64:
        return "INT64";
    case TypeId::UINT8:
        return "UINT8";
    case TypeId::UINT16:
        return "UINT16";
    case TypeId::UINT32:
        return "UINT32";
    case TypeId::UINT64:
        return "UINT64";
    case TypeId::FLOAT32:
        return "FP32";
    case TypeId::FLOAT64:
        return "FP64";
    case TypeId::BOOL8:
        return "BOOL";
    case TypeId::EMPTY:
    case TypeId::NUM_TYPE_IDS:
    default:
        throw std::runtime_error("Not supported");
    }
}

// From cudf
DType DType::from_cudf(cudf::type_id tid)
{
    switch (tid)
    {
    case cudf::type_id::INT8:
        return DType(TypeId::INT8);
    case cudf::type_id::INT16:
        return DType(TypeId::INT16);
    case cudf::type_id::INT32:
        return DType(TypeId::INT32);
    case cudf::type_id::INT64:
        return DType(TypeId::INT64);
    case cudf::type_id::UINT8:
        return DType(TypeId::UINT8);
    case cudf::type_id::UINT16:
        return DType(TypeId::UINT16);
    case cudf::type_id::UINT32:
        return DType(TypeId::UINT32);
    case cudf::type_id::UINT64:
        return DType(TypeId::UINT64);
    case cudf::type_id::FLOAT32:
        return DType(TypeId::FLOAT32);
    case cudf::type_id::FLOAT64:
        return DType(TypeId::FLOAT64);
    case cudf::type_id::BOOL8:
        return DType(TypeId::BOOL8);
    case cudf::type_id::EMPTY:
    case cudf::type_id::NUM_TYPE_IDS:
    default:
        throw std::runtime_error("Not supported");
    }
}

// From triton
DType DType::from_triton(const std::string& type_str)
{
    if (type_str == "INT8")
    {
        return DType(TypeId::INT8);
    }
    else if (type_str == "INT16")
    {
        return DType(TypeId::INT16);
    }
    else if (type_str == "INT32")
    {
        return DType(TypeId::INT32);
    }
    else if (type_str == "INT64")
    {
        return DType(TypeId::INT64);
    }
    else if (type_str == "UINT8")
    {
        return DType(TypeId::UINT8);
    }
    else if (type_str == "UINT16")
    {
        return DType(TypeId::UINT16);
    }
    else if (type_str == "UINT32")
    {
        return DType(TypeId::UINT32);
    }
    else if (type_str == "UINT64")
    {
        return DType(TypeId::UINT64);
    }
    else if (type_str == "FP32")
    {
        return DType(TypeId::FLOAT32);
    }
    else if (type_str == "FP64")
    {
        return DType(TypeId::FLOAT64);
    }
    else if (type_str == "BOOL")
    {
        return DType(TypeId::BOOL8);
    }
    else
    {
        throw std::runtime_error("Not supported");
    }
}

}  // namespace morpheus
